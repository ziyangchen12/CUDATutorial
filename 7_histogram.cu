#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

//2.386ms
__global__ void histgram(int *hist_data, int *bin_data)
{
    int gtid = blockIdx.x * blockDim.x + threadIdx.x;
    /*error*/
    // bin_data[hist_data[gtid]]++;
    /*right*/
    atomicAdd(&bin_data[hist_data[gtid]], 1);//性能延迟不好的原因，会造成bank conflict，因为val在这个数组histgram中有很多，每一个单线程计算全局内存中的若干个值val，那么就会有多个线程访问相同的val，
//一个疑问点是多个相同的val在smem中是只存储一个吗？还是存放在不同的位置呢？
//答案：这里是对于bindata这个存储位置的bank conflict吧，一个wrap中的线程有可能都会访问同一个index的bin_data[index]导致了bank conflict？

}

bool CheckResult(int *out, int* groudtruth, int N){
    for (int i = 0; i < N; i++){
        if (out[i] != groudtruth[i]) {
            return false;
        }
    }
    return true;
}

int main(){
    float milliseconds = 0;
    const int N = 25600000;
    int *hist = (int *)malloc(N * sizeof(int));
    int *bin = (int *)malloc(256 * sizeof(int));
    int *bin_data;
    int *hist_data;
    hipMalloc((void **)&bin_data, 256 * sizeof(int));
    hipMalloc((void **)&hist_data, N * sizeof(int));

    for(int i = 0; i < N; i++){
        hist[i] = i % 256;
    }

    int *groudtruth = (int *)malloc(256 * sizeof(int));;
    for(int j = 0; j < 256; j++){
        groudtruth[j] = 100000;
    }

    hipMemcpy(hist_data, hist, N * sizeof(int), hipMemcpyHostToDevice);
    hipSetDevice(0);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    const int blockSize = 256;
    int GridSize = std::min((N + 256 - 1) / 256, deviceProp.maxGridSize[0]);
    dim3 Grid(GridSize);
    dim3 Block(blockSize);
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    histgram<<<Grid, Block>>>(hist_data, bin_data);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);

    hipMemcpy(bin, bin_data, 256 * sizeof(int), hipMemcpyDeviceToHost);
    bool is_right = CheckResult(bin, groudtruth, 256);
    if(is_right) {
        printf("the ans is right\n");
    } else {
        printf("the ans is wrong\n");
        for(int i = 0; i < 256; i++){
            printf("%lf ", bin[i]);
        }
        printf("\n");
    }
    printf("histogram latency = %f ms\n", milliseconds);    

    hipFree(bin_data);
    hipFree(hist_data);
    free(bin);
    free(hist);
}
