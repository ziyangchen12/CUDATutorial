#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

//注释的这些内容需要在ampre 级别的gpu才可以进行运算，google colab只有t4级别的gpu可以进行运算
template <typename T, int Size>
struct alignas(sizeof(T) * Size) AlignedVector {
  T val[Size];
  __host__ __device__ inline const T& operator[](int i) const { return val[i]; }
  __host__ __device__ inline T& operator[](int i) { return val[i]; }
};

__device__ float TanhApprox(float x) {
  //float r;
  //asm("tanh.approx.f32 %0,%1; \n\t" : "=f"(r) : "f"(x));
  //return r;
  return tanhf(x);
}

template<typename T>
struct GeluFunctor {
  static constexpr T alpha = static_cast<T>(0.7978845608028654);
  static constexpr T beta = static_cast<T>(0.044714998453855515);

  __device__ GeluFunctor() {};

  __device__ T operator()(T x) const {
    const T half = static_cast<T>(0.5);
    const T one = static_cast<T>(1);
    const T tanh_in = alpha * (x + beta * x * x * x);
    return half * x * (one + tanh(tanh_in));
  }
};

template<>
struct GeluFunctor<half> {
  static constexpr float alpha = GeluFunctor<float>::alpha;
  static constexpr float beta = GeluFunctor<float>::beta;
  GeluFunctor<float> float_functor;

  __device__ GeluFunctor() {};

  __device__ half operator()(const half x) const {
    // Note: when you have ampere GPU, you can enable the line45-50 method to get performance improvement by half intrinsic instead of static_cast half to fp32.
    //const float tanh_in =
    //    __half2float(__float2half_rn(alpha) * (x + __float2half_rn(beta) * x * x * x));
    //const float tanh_out = TanhApprox(tanh_in);
    //return __float2half_rn(0.5f) * x * (__float2half_rn(1.0f) + __float2half_rn(tanh_out));
    // Note: half to float will lose performance using static_cast, because static_cast will be compiled to more instructions than half intrinsic,
    // so you should better use half intrinsic when you have ampere GPU, you can enable 44-47 line
    return static_cast<half>(float_functor(static_cast<float>(x)));
  }
  // Note: when you have ampere GPU, you can enable the "apply2" method to get performance improvement by half2 intrinsic.
  //__device__ void Apply2(half* y, const half* x) const {
    //const half2 x2 = *(reinterpret_cast<const half2*>(x));
    //const float2 tanh_in = __half22float2(
     //   __hmul2(__float2half2_rn(alpha),
      //          __hadd2(x2, __hmul2(__hmul2(__hmul2(__float2half2_rn(beta), x2), x2), x2))));
    //float2 tanh_out;
    //tanh_out.x = TanhApprox(tanh_in.x);
    //tanh_out.y = TanhApprox(tanh_in.y);
    //const half2 y2 = __hmul2(__hmul2(__hmul2(__float2half2_rn(0.5F), x2),
    //                                 __hadd2(__float2half2_rn(1.0F), __float22half2_rn(tanh_out))));
    //*reinterpret_cast<half2*>(y) = y2;
  //}
};
//知识的补充：
/*指针做偏移操作的时候是根据指针的类型来做出移位判断，比如这个例子里面指针是整形，所以每次偏移的时候是按照整形的长度（4字节，32位）偏移，如果指针是char类型，则每次加1是加1个字节。
//指针自增每次增加的增加量=sizof（你所用的数据类型)
//比如
//char型指针自增的话地址加1
//int型一般都是4字节，所以自增会加4.*/


template <int VecSize>
__global__ void FP16GeluCUDAKernel(const __half* x,
                                                 __half* y,
                                                 int n) {
  int offset =
      static_cast<int>(threadIdx.x + blockIdx.x * blockDim.x) * VecSize;
  int stride = static_cast<int>(blockDim.x * gridDim.x) * VecSize;//数据量不能被所有线程整除的时候，也就是一部分数据只能被部分数据处理
  GeluFunctor<half> gelu_fwd;
  __half y_reg[VecSize];
  for (; offset < n; offset += stride) {
    using ArrT = AlignedVector<__half, VecSize>;
    const ArrT* in_arr = reinterpret_cast<const ArrT*>(x + offset);
    // ArrT* out_arr = reinterpret_cast<const ArrT*>(y + offset);
    const __half* in = reinterpret_cast<const __half*>(in_arr);
    // __half* out = reinterpret_cast<const __half*>(out_arr);
//两次指针强转的原因：1、是为了进行内存对齐吧，加快内存存取速度 2、转变回来的原因是如果原来是ArrT类型的，每次加一就是加上ArrT的长度了，如果VecSize不是1，那么in[0]就不是一个数字了，而是一个数组？
    if (VecSize == 1){
        y_reg[0] = gelu_fwd(in[0]);
    } else {
      // Note: when you have ampere GPU, you can enable the "apply2" method to get performance improvement by half2 intrinsic.
      //for (int i = 0; i < VecSize; i+=2) {
      //gelu_fwd.apply2(y + offset, in[i]);
        for (int i = 0; i < VecSize; i++) {
            y_reg[i] = gelu_fwd(in[i]);
        }
    }
    *reinterpret_cast<ArrT*>(y + offset) = *reinterpret_cast<ArrT*>(y_reg);
  }
}

int main() {
    int n = 1000;
    
    __half *x = new __half[n];
    __half *y = new __half[n];
    for (int i = 0; i < n; i++)
    {
      x[i] = (__half)(i);
    }
    __half * d_x, *d_y;
    hipMalloc((void **)&d_x, n * sizeof(__half));
    hipMalloc((void **)&d_y, n * sizeof(__half));
    hipMemcpy(d_x, x, sizeof(__half) * n, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(__half) * n, hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);

    auto is_aligned = [](const void* p, int alignment) {
        return reinterpret_cast<uintptr_t>(p) % alignment == 0;
    };
                                                                      
    constexpr auto kAlignment = alignof(AlignedVector<__half, 8>); 
    // Note: when you have ampere GPU, you can enable the 122-124 line to get performance improvement by half2 intrinsic.
    if (n % 8 == 0 && is_aligned(x, kAlignment) && is_aligned(y, kAlignment)) {                                          
      int thread = std::min<int>(512, deviceProp.maxThreadsPerBlock); 
      //int block = (n / 8 + thread - 1) / thread;                  
      //block = std::min<int>(block, deviceProp.maxGridSize[0]);                                  
      //FP16GeluCUDAKernel<8, true><<<block, thread>>>(x, y, n);  
      int block = (n + thread - 1) / thread;                  
      block = std::min<int>(block, deviceProp.maxGridSize[0]);                                  
      FP16GeluCUDAKernel<1><<<block, thread>>>(d_x, d_y, n);                      
      hipMemcpy(y, d_y, sizeof(__half) * n, hipMemcpyDeviceToHost);                                                          
    }   
    printf("pass");
    delete x;
    x = nullptr;
    delete y;
    y = nullptr;
    hipFree(d_x);
    hipFree(d_y);
}
